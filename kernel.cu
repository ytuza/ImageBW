#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "image_cuda.h"

__global__ void image_bound(unsigned char* image, int Channels, int xm, int ym) {

	int x = blockIdx.x;
	int y = blockIdx.y;

	int idx = (x + y * gridDim.x) * Channels;

	int sumR = 0, sumG = 0, sumB = 0;

	int idx0 = (x+1 + y * gridDim.x) * Channels;
	int idx1 = (x-1 + y * gridDim.x) * Channels;
	int idx2 = (x + (y+1) * gridDim.x) * Channels;
	int idx3 = (x + (y-1) * gridDim.x) * Channels;
	int idx4 = (x+1 + (y+1) * gridDim.x) * Channels;
	int idx5 = (x+1 + (y-1) * gridDim.x) * Channels;
	int idx6 = (x-1 + (y + 1) * gridDim.x) * Channels;
	int idx7 = (x-1 + (y - 1) * gridDim.x) * Channels;



	if (x == 0) {
		if (y == 0) {
			sumR = image[idx2] + image[idx4] + image[idx0];
			sumG = image[idx2+1] + image[idx4+1] + image[idx0+1];
			sumB = image[idx2+2] + image[idx4+2] + image[idx0+2];

			sumR = sumR / 3;
			sumG = sumG / 3;
			sumB = sumB / 3;
		}
		else if (y == ym-1) {
			sumR = image[idx3] + image[idx5] + image[idx0];
			sumG = image[idx3 + 1] + image[idx5 + 1] + image[idx0 + 1];
			sumB = image[idx3 + 2] + image[idx5 + 2] + image[idx0 + 2];
			sumR = sumR / 3;
			sumG = sumG / 3;
			sumB = sumB / 3;
		}
		else {
			sumR = image[idx2] + image[idx4] + image[idx0] + image[idx3]+ image[idx5];
			sumG = image[idx2 + 1] + image[idx4 + 1] + image[idx0 + 1] + image[idx3+1] + image[idx5+1];
			sumB = image[idx2 + 2] + image[idx4 + 2] + image[idx0 + 2] + image[idx3+2] + image[idx5+2];
			sumR = sumR / 5;
			sumG = sumG / 5;
			sumB = sumB / 5;
		}
	}else if (x == xm-1) {
		if (y == ym-1) {
			sumR = image[idx1] + image[idx7] + image[idx3];
			sumG = image[idx1 + 1] + image[idx7 + 1] + image[idx3 + 1];
			sumB = image[idx1 + 2] + image[idx7 + 2] + image[idx3 + 2];
			sumR = sumR / 3;
			sumG = sumG / 3;
			sumB = sumB / 3;
		}
		else if (y == 0) {
			sumR = image[idx1] + image[idx6] + image[idx2];
			sumG = image[idx1 + 1] + image[idx6 + 1] + image[idx2 + 1];
			sumB = image[idx1 + 2] + image[idx6 + 2] + image[idx2 + 2];
			sumR = sumR / 3;
			sumG = sumG / 3;
			sumB = sumB / 3;
		}
		else {
			sumR = image[idx2] + image[idx6] + image[idx1] + image[idx3] + image[idx7];
			sumG = image[idx2 + 1] + image[idx6 + 1] + image[idx1 + 1] + image[idx3 + 1] + image[idx7 + 1];
			sumB = image[idx2 + 2] + image[idx6 + 2] + image[idx1 + 2] + image[idx3 + 2] + image[idx7 + 2];
			sumR = sumR / 5;
			sumG = sumG / 5;
			sumB = sumB / 5;
		}
	}
	else if (y == ym - 1) {
		sumR = image[idx1] + image[idx7] + image[idx3] + image[idx5] + image[idx0];
		sumG = image[idx1 + 1] + image[idx7 + 1] + image[idx3 + 1] + image[idx5 + 1] + image[idx0 + 1];
		sumB = image[idx1 + 2] + image[idx7 + 2] + image[idx3 + 2] + image[idx5 + 2] + image[idx0 + 2];
		sumR = sumR / 5;
		sumG = sumG / 5;
		sumB = sumB / 5;
	}
	else if (y == 0) {
		sumR = image[idx1] + image[idx6] + image[idx2] + image[idx4] + image[idx0];
		sumG = image[idx1 + 1] + image[idx6+ 1] + image[idx2 + 1] + image[idx4 + 1] + image[idx0 + 1];
		sumB = image[idx1 + 2] + image[idx6 + 2] + image[idx2 + 2] + image[idx4 + 2] + image[idx0 + 2];
		sumR = sumR / 5;
		sumG = sumG / 5;
		sumB = sumB / 5;
	}
	else {
		sumR = image[idx1] + image[idx6] + image[idx2] + image[idx4] + image[idx0] + image[idx7] + image[idx3] + image[idx5];
		sumG = image[idx1 + 1] + image[idx6 + 1] + image[idx2 + 1] + image[idx4 + 1] + image[idx0 + 1] + image[idx7 + 1] + image[idx3 + 1] + image[idx5 + 1];
		sumB = image[idx1 + 2] + image[idx6 + 2] + image[idx2 + 2] + image[idx4 + 2] + image[idx0 + 2] + image[idx7 + 2] + image[idx3 + 2] + image[idx5 + 2];
		sumR = sumR / 8;
		sumG = sumG / 8;
		sumB = sumB / 8;
	}



	image[idx] = sumR;
	image[idx + 1] = sumG;
	image[idx + 2] = sumB;

}

__global__ void image_Inv(unsigned char* image, int Channels) {

	int x = blockIdx.x;
	int y = blockIdx.y;

	int idx = (x + y * gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++) {
		image[idx + i] = 255 - image[idx + i];
	}
	
}

__global__ void image_BW(unsigned char * image, int Channels) {

	int x = blockIdx.x;
	int y = blockIdx.y;

	int idx = (x + y * gridDim.x) * Channels;

	int valgrey = (int)((0.21 * image[idx]) + (0.72 * image[idx + 1]) + (0.07 * image[idx + 2]));
	image[idx] = valgrey;
	image[idx+1] = valgrey;
	image[idx+2] = valgrey;

}


void conv_CUDA(unsigned char* input_image, int Height, int Width, int Channels) {
	unsigned char* pt_image = NULL;
	hipMalloc((void**)&pt_image, Height*Width*Channels);

	hipMemcpy(pt_image, input_image,  Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid(Width, Height);

	//image_BW <<<  Grid, 1 >>> (pt_image, Channels);
	image_bound << <  Grid, 1 >> > (pt_image, Channels, Width, Height);

	hipMemcpy(input_image, pt_image, Height * Width * Channels, hipMemcpyDeviceToHost);
	
	hipFree(pt_image);
}

